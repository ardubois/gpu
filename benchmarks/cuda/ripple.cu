
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <malloc.h>
#include <math.h>
#define _bitsperpixel 32
#define _planes 1
#define _compression 0

#define _xpixelpermeter 0x13B //0x130B //2835 , 72 DPI
#define _ypixelpermeter 0x13B//0x130B //2835 , 72 DPI
#define pixel 0xFF
#pragma pack(push,1)
typedef struct{
    uint8_t signature[2];
    uint32_t filesize;
    uint32_t reserved;
    uint32_t fileoffset_to_pixelarray;
} fileheader;
typedef struct{
    uint32_t dibheadersize;
    uint32_t width;
    uint32_t height;
    uint16_t planes;
    uint16_t bitsperpixel;
    uint32_t compression;
    uint32_t imagesize;
    uint32_t ypixelpermeter;
    uint32_t xpixelpermeter;
    uint32_t numcolorspallette;
    uint32_t mostimpcolor;
} bitmapinfoheader;
typedef struct {
    fileheader fileheader;
    bitmapinfoheader bitmapinfoheader;
} bitmap;
#pragma pack(pop)
__global__
void kernel(uint8_t *ptr, uint32_t DIM, int ticks ){
   int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y  = threadIdx.y + blockIdx.y * blockDim.y;
    int  offset  = x + y * blockDim.x * gridDim.x;
            
            float fx = 0.5 *  x - DIM/15;
	    float fy = 0.5* y - DIM/15;
	    float d = sqrtf( fx * fx + fy * fy );
	    unsigned char grey = (unsigned char)(128.0f + 127.0f * cos(d/10.0f - ticks/7.0f) / (d/10.0f + 1.0f));
		ptr[offset*4 + 0] = grey;
		ptr[offset*4 + 1] = grey;
		ptr[offset*4 + 2] = grey;
		ptr[offset*4 + 3] = 255;
        
 }
 
 void genBpm (uint32_t height, uint32_t width, uint8_t *pixelbuffer) {
    uint32_t pixelbytesize = height*width*_bitsperpixel/8;
    uint32_t  _filesize =pixelbytesize+sizeof(bitmap);
    FILE *fp = fopen("test.bmp","wb");
    bitmap *pbitmap  = (bitmap*)calloc(1,sizeof(bitmap));


    //strcpy(pbitmap->fileheader.signature,"BM");
    pbitmap->fileheader.signature[0] = 'B';
    pbitmap->fileheader.signature[1] = 'M';
    pbitmap->fileheader.filesize = _filesize;
    pbitmap->fileheader.fileoffset_to_pixelarray = sizeof(bitmap);
    pbitmap->bitmapinfoheader.dibheadersize =sizeof(bitmapinfoheader);
    pbitmap->bitmapinfoheader.width = width;
    pbitmap->bitmapinfoheader.height = height;
    pbitmap->bitmapinfoheader.planes = _planes;
    pbitmap->bitmapinfoheader.bitsperpixel = _bitsperpixel;
    pbitmap->bitmapinfoheader.compression = _compression;
    pbitmap->bitmapinfoheader.imagesize = pixelbytesize;
    pbitmap->bitmapinfoheader.ypixelpermeter = _ypixelpermeter ;
    pbitmap->bitmapinfoheader.xpixelpermeter = _xpixelpermeter ;
    pbitmap->bitmapinfoheader.numcolorspallette = 0;
    fwrite (pbitmap, 1, sizeof(bitmap),fp);
    //memset(pixelbuffer,pixel,pixelbytesize);
    fwrite(pixelbuffer,1,pixelbytesize,fp);
    fclose(fp);
    free(pbitmap);
    free(pixelbuffer);
}


int main( void ) {
   
   uint32_t height = 1000;
    uint32_t width  = 1000;
    uint32_t DIM = 1000;
    hipError_t j_error;
    
    uint32_t pixelbytesize=  height*width*_bitsperpixel/8;
     
    //printf(" pixel byte size %lu\n",pixelbytesize);
   
     uint8_t *h_pixelbuffer = (uint8_t*)malloc(pixelbytesize);
     uint8_t *d_pixelbuffer;

     ////////
    hipMalloc( (void**)&d_pixelbuffer, pixelbytesize);
    j_error = hipGetLastError();
    if(j_error != hipSuccess) printf("Error 1: %s\n", hipGetErrorString(j_error));
    ////////

    
    ////////////////////
    dim3 grid(DIM,DIM);

    kernel<<<grid, 1>>>(d_pixelbuffer,height,10); // compute interbody forces
    j_error = hipGetLastError();
    if(j_error != hipSuccess) printf("Error 3: %s\n", hipGetErrorString(j_error));
  ////////



    hipMemcpy(h_pixelbuffer, d_pixelbuffer, pixelbytesize, hipMemcpyDeviceToHost); // return results 
    j_error = hipGetLastError();
    if(j_error != hipSuccess) printf("Error 7: %s\n", hipGetErrorString(j_error));

    
    genBpm(height,width,h_pixelbuffer);
    
}
