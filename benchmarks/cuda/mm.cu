
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void gpu_mm(float *a,float *b, float *c, int m, int n, int k)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if( col < k && row < m) 
    {
        for(int i = 0; i < n; i++) 
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
} 

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main(int argc, char const *argv[])
{
    int m = 1000;
    int block_size = 128;

    float *a = (float*) malloc(m*m*sizeof(float));
    float *b = (float*) malloc(m*m*sizeof(float));
    float *c = (float*) malloc(m*m*sizeof(float));

    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < m; ++j) {
            a[i * m + j] = 2.0;
        }
    }

    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < m; ++j) {
            b[i * m + j] = 3.0;
        }
    }

    float *d_a, *d_b, *d_c;


    hipMalloc((void **) &d_a, sizeof(float)*m*m);
    hipMalloc((void **) &d_b, sizeof(float)*m*m);
    hipMalloc((void **) &d_c, sizeof(float)*m*m);

    // copy matrix A and B from host to device memory
    hipMemcpy(d_a, a, sizeof(float)*m*m, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float)*m*m, hipMemcpyHostToDevice);

    int grid_rows = (m + block_size - 1) / block_size;
    int grid_cols = (m + block_size - 1) / block_size;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(block_size, block_size);
   
    gpu_mm<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, m,m,m);    

    hipMemcpy(c, d_c, sizeof(float)*m*m, hipMemcpyDeviceToHost);

    checkElementsAre(5.0,c,m*m);

    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

}
    